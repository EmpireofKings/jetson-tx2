#include "hip/hip_runtime.h"
__global__ void gpu_matrix_mult_one(int *a, int *b, int *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y; // get the row
    int col = blockIdx.x * blockDim.x + threadIdx.x; // get the column
    int sum = 0; // initialize the sum

    if( col < k && row < m) // check to make sure that the thread needs to compute
    {
        for(int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

__global__ void gpu_matrix_mult_two(float *d_M, float *d_N, float *d_P, int m, int n, int k){

    // shared memory for tiling
    __shared__ float Mds [TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds [TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    // recall that TILE_WIDTH = blockDim
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float pval = 0;

    // this loop is iterating through cols of M and rows of N
    // recall that n is the shared inner dimension, that's why we're using it
    // to define our loop size
    for (int ph = 0; ph < ceil(n / (float) TILE_WIDTH); ph++){
        // boundary check for shared Mds
        if (row < k && ph * TILE_WIDTH + tx < m){
            // saving tile from M
            /* indexing thought exercise:
             * "row * k" gets us to our desired row in M
             * adding "ph * TILE_WIDTH" moves our tile over to the desired tile location
             * adding "tx" moves us to the desired location within the tile
             * */
            Mds[ty][tx] = d_M[row * k + ph * TILE_WIDTH + tx];
        }
        // boundary check
        if (ph*TILE_WIDTH + ty < k && col < m){
            // saving tile from N
            /* indexing thought exercise:
             * "ph * TILE_WIDTH" moves the tile "down" to the desired location
             * adding "ty" gets us to the desired location within the tile
             * multiplying by "k" does the magic (remember row major order)
             * adding col moves the tile to the desired column*/
            Nds[ty][tx] = d_N[(ph * TILE_WIDTH + ty) * k + col];
        }

        __syncthreads();  // execution barrier

        for (int j = 0; j < TILE_WIDTH; j++){
            // performing part of inner product
            pval += Mds[ty][j] * Nds[j][tx];
        }

        __syncthreads();
    }
    if (row < k && col < m){
        d_P[row * k + col] = pval;
    }
}
