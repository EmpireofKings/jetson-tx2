
#include <hip/hip_runtime.h>
#include <stdio.h>


/*
The "__global__" tag tells nvcc that the function will execute on the device
but will be called from the host. Notice that we must use pointers!
*/
__global__
void add_int( int *a, int *b, int *c){
  *c = *a + *b;
  printf("blockIdx: %d\n",blockIdx.x);
}

// Main program
int main(void){

  //host memory != device memory, must allocate differently
  //device pointers point to GPU Memory
  //host pointers point to CPU memory
  int a, b, c;                //host copies
  int *dev_a, *dev_b, *dev_c; //device copies
  int size = sizeof( int );   //size of an interger

  //allocate space on device
  hipMalloc( (void**)&dev_a, size );
  hipMalloc( (void**)&dev_b, size );
  hipMalloc( (void**)&dev_c, size );

  a = 2; //storing values in host
  b = 7;

  // now we need the values to be copied to the device
  hipMemcpy( dev_a, &a, size, hipMemcpyHostToDevice );
  hipMemcpy( dev_b, &b, size, hipMemcpyHostToDevice );

  // launch the add_int kernel on the GPU
  add_int<<<3,1>>>(dev_a, dev_b, dev_c);

  //now we want the values back on the CPU
  hipMemcpy( &c, dev_c, size, hipMemcpyDeviceToHost );

  printf("C: %d\n",c);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);


  // your basic hello world program
  printf("Hello, World!\n");
  return 0;
}
