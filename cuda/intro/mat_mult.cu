
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>


void native_mat_mult(int *a, int *b, int *result, int m, int n, int k)
{
	for (int i = 0; i < m; i++)
	{
		for (int j = 0; j < n; j++)
		{
			int tmp = 0.0;
			for (int h = 0; h < n; h++)
			{
				tmp += a[i * n + h] * b[h * k + j];
			}
		result[i * k + j] = tmp;
		}
	}

}


int main(int argc, char const *argv[])
{
	int m, n, k;
	printf("Input m n k::\n");
	scanf("%d %d %d", &m, &n, &k);

	int *cpu_a, *cpu_b, *cpu_result;

	hipHostMalloc((void **) &cpu_a, sizeof(int)*m*n, hipHostMallocDefault);
	hipHostMalloc((void **) &cpu_b, sizeof(int)*m*n, hipHostMallocDefault);
	hipHostMalloc((void **) &cpu_result, sizeof(int)*m*n, hipHostMallocDefault);

	for (int i = 0; i < m; i++)
	{
		for (int j = 0; j < n; j++)
		{
			cpu_a[i * n + j] = rand () % 1024;
		}
	}

        for (int i = 0; i < n; i++)
        {
                for (int j = 0; j < k; j++)
                {
                        cpu_b[n * k + j] = rand () % 1024;
                }
        }

	float cpu_time;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//cpu
	hipEventRecord(start,0);

	native_mat_mult(cpu_a, cpu_b, cpu_result, m, n, k);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&cpu_time, start, stop);
	
	printf("Time on mat mult of %d%d * %d%d on CPU: %f ms.\n\n", m, n, n, k, cpu_time);

	hipHostFree(cpu_a);
	hipHostFree(cpu_b);
	hipHostFree(cpu_result);
	return 0;
}
