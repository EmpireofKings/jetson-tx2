#include "hip/hip_runtime.h"
//
// Created by matt on 3/18/18.
//

#include <iostream>
#include <string>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>


// -----------------CONSTANT MEMORY-------------------------------------
#define BLOCK_SIZE 32

// size of mask dimensions
#define MASK_SIZE 5


// error checking macro
// wrap api calls with this
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


// constant memory on host to save kernel for fast access
__constant__ float M[MASK_SIZE * MASK_SIZE];

// -------------------------------------------- GPU KERNELS ------------------------------------------------------------

// CONVOLUTION
__global__ void gpu_convolve2d(unsigned char* Pout, unsigned char* Pin,
                               int width, int height) {

    // index into the image
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = y * width + x;

    // commonly used operation so only do it once and store it in register
    unsigned char n = MASK_SIZE / 2;

    // shared memory to store tile
    __shared__ float Pout_ds[BLOCK_SIZE * BLOCK_SIZE];

    // load tiles into shared memory
    if (x < width && y < height) {
//        Pout_ds[threadIdx.y * blockDim.y + threadIdx.x] = Pin[i];
        Pout_ds[threadIdx.y * blockDim.x + threadIdx.x] = float(Pin[i]);
    }

    __syncthreads();  // execution barrier


    // remember that we are tiling with blocks size equal to tile size
    // we need to create bounds for the tiles

    float P_sum = 0.0;

    // loop through rows of mask
    for (int j = -n; j < n + 1; j++) {
        // loop through columns of mask
        for (int k = n; k < n + 1; k++) {

            // get vars, delete if too many registers being used.
            // index into the tile
            int x_index = threadIdx.x + k;
            int y_index = threadIdx.y + j;

            // check that we are in bounds of the shared memory
            if (x_index > -1 &&
                    x_index < blockDim.x &&
                    y_index > -1 &&
                    y_index < blockDim.y) {

                // accumulate with weight from mask "M"
                P_sum += Pout_ds[y_index * blockDim.x + x_index] * M[(n + j)*MASK_SIZE + n + k];
            }
                // check that we're within bounds of the image, otherwise just make the pixel zero
            else if(x + k > -1 &&
                    x + k < width &&
                    y + j > -1 &&
                    y + j < height){

                // grab from global memory (probably cached since other threads are likely pulling this data)
                // accumulate wieth weight from mask "M"
                P_sum += float(Pin[(y+j)*width + x + k]) * M[(n + j)*MASK_SIZE + n + k];
            }
        }
    }

    // make sure that the pixel is within the range of the picture
    if (x < width && y < height) {
        Pout[i] = (unsigned char) P_sum;
    }
}


// ------------------------------------more naive version of the blur --------------------------------------------------
__global__ void gpu_blur(unsigned char* Pout, unsigned char* Pin, int width, int height){
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int k_size = 3;

    if (col < width && row < height){
        int pixVal = 0;
        int pixels = 0;

        for(int blurRow = -k_size; blurRow < k_size+1; blurRow++){
            for(int blurCol = -k_size; blurCol < k_size+1; blurCol++){
                int curRow = row + blurRow;
                int curCol = col + blurCol;

                if (curRow > -1 && curRow < height && curCol > -1 && curCol < width){
                    pixVal += Pin[curRow * width + curCol];
                    pixels++;
                }
            }
        }

        Pout[row * width + col] = (unsigned char) (pixVal / pixels);
    }
}

// KERNEL TO PLAY WITH
__global__ void gpu_test(unsigned char* Pout, unsigned char* Pin, int width, int height) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int i = row * width + col;

    if (row < height && col < width) {
        Pout[i] = Pin[i];
    }

}

// --------------------------------------cpu functions------------------------------------------------------------------

void cpu_blur(cv::Mat src, cv::Mat dst){
    int num_rows = src.rows;
    int num_cols = src.cols;

    // size of kernel (i.e. 3 = -3 -> 3 so size of 7 px)
    unsigned char ksize = 3;
    for(int i=ksize; i<num_rows-ksize; i++) {
        for (int j = ksize; j < num_cols-ksize; j++) {
            unsigned int pixsum = 0;
            unsigned int numpx = 0;
            // You can now access the pixel value with cv::Vec3b
            for (int k=-ksize; k < ksize+1; k++){
                for (int l=-ksize; l < ksize+1; l++) {
                    numpx++;
                    pixsum += (unsigned int) src.at<uchar>(i + k, j + l);
                }
            }
            dst.at<uchar>(i,j) = (unsigned char) (pixsum / numpx);
        }
    }
}


// -----------MAIN-------------------MAIN-----------------MAIN----------------MAIN-------------MAIN---------------------

int main(){

    float bottom = 7.0;
    // ------------------------------- create a mask for convolving! ---------------------------------------------------
    float convolve_mask5x5[25] = {
            1.f/bottom, 1.f/bottom, 1.f/bottom, 1.f/bottom, 1.f/bottom,
            1.f/bottom, 2.f/bottom, 2.f/bottom, 2.f/bottom, 1.f/bottom,
            1.f/bottom, 2.f/bottom, 3.f/bottom, 2.f/bottom, 1.f/bottom,
            1.f/bottom, 2.f/bottom, 2.f/bottom, 2.f/bottom, 1.f/bottom,
            1.f/bottom, 1.f/bottom, 1.f/bottom, 1.f/bottom, 1.f/bottom
    };

    // copy the mask from the host to the device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(M), convolve_mask5x5, MASK_SIZE * MASK_SIZE * sizeof(float));

    // ----------------getting the camera stream-----------------------------------
    const char* gst;
    gst = "nvcamerasrc ! video/x-raw(memory:NVMM), width=(int)1280, height=(int)720, format=(string)I420, framerate=(fraction)30/1 ! \
                nvvidconv flip-method=0 ! video/x-raw, format=(string)BGRx ! \
                videoconvert ! video/x-raw, format=(string)BGR ! \
                appsink";

    // open the camera with the gst string
    // If you want to use a webcam, change this to cap(1) (at least on the jetson tx2)
    cv::VideoCapture cap(gst);

    // error handling
    if(!cap.isOpened()) {
        printf("Failed To Open Camera");
        return -1;
    }


    // get info about the picture
    unsigned int width = cap.get(CV_CAP_PROP_FRAME_WIDTH);
    unsigned int height = cap.get(CV_CAP_PROP_FRAME_HEIGHT);
    unsigned int pixels = width * height;

    // print info about the picture
    printf("Frame Size: %d x %d\n %d pixels\n", width, height, pixels);

    // create opnecv Mat container for image
    cv::Mat frame_in;

    // grab the first frame, this will also automatically size the container
    cap >> frame_in;

    // create windows for different filters
    cv::namedWindow("Source");  // this window will hold the source image
    cv::namedWindow("Result");  // this window will hold the resultant image


    // ------------------- ALLOCATE GPU MEMORY ------------------------------------
    unsigned char *d_Pin, *d_Pout;  // init pointers
    gpuErrchk( hipMalloc((void **) &d_Pin, sizeof(unsigned char)*pixels) );  // allocate space on device for input
    gpuErrchk( hipMalloc((void **) &d_Pout, sizeof(unsigned char)*pixels) ); // allocate space on device for output


    // ------------------- ALLOCATE CPU MEMORY -------------------------------------
    // note that input image has space allocated already by frame_in
    unsigned char *h_Pout = (unsigned char *) malloc(pixels*sizeof(char)); // allocate space for output


    // ----------------------------GPU POLLING--------------------------------------------------------------------------
    // todo - use this info to optomize the program for any GPU
    int dev_count;
    hipGetDeviceCount(&dev_count);

    hipDeviceProp_t dev_prop;
    for (int i = 0; i < dev_count; i++){
        hipGetDeviceProperties(&dev_prop, i);
        printf("maxThreadsPerBlock: %d\n",dev_prop.maxThreadsPerBlock);
        printf("multiProcessorCount: %d\n",dev_prop.multiProcessorCount);
        printf("wrapSize: %d\n",dev_prop.warpSize);
        printf("regsPerBlock: %d\n", dev_prop.regsPerBlock);
        printf("maxThreadsPerMultiProcessor: %d\n",dev_prop.maxThreadsPerMultiProcessor);
    }

    // -----------------------------BLOCKS AND THREADS------------------------------------------------------------------
    int numThreadsX = BLOCK_SIZE;
    dim3 numThreads(numThreadsX, numThreadsX);
    int numBlocksX = (width)/numThreadsX;
    int numBlocksY = (height)/numThreadsX;
    dim3 numBlocks(numBlocksX, numBlocksY);

    printf("\nnumBlocksX: %d\n",numBlocksX);
    printf("numBlocksY: %d\n",numBlocksY);
    printf("numBlocks: %d\n",numBlocks.x * numBlocks.y);
    printf("threads per block: %d\n", numThreads.x * numThreads.y);
    printf("total threads: %d\n",(numBlocks.x * numBlocks.y)*numThreads.x*numThreads.y);


    printf("\n\n\n\n===========================================================\n"
                   " TOGGLE WITH \"1\", \"2\", and \"3\" on your keyboard!\n"
                   "\"q\" to quit\n"
                   "\"c\" for CPU\n"
                   "===========================================================\n\n\n\n");

    // init keystroke variables
    char keystroke = '1';  // default keystroke (GPU)

    while(1){

        //---------------------Keyboard Input Handling------------------------------------------------------------------
        // get the keystroke
        char tmp_key = (char) cv::waitKey(1);

        // check if a key has been pressed
        if (tmp_key != 255){
            keystroke = tmp_key;
            tmp_key = 255;
        }


        //---------------------Read in the image------------------------------------------------------------------------
        // direct image from camera to cv::Mat object
        cap >> frame_in;

        // convert color to bw
        cv::cvtColor(frame_in, frame_in, CV_BGR2GRAY);

        switch(keystroke) {
            case 'q' : {
                // ------------------QUIT CASE--------------------------------------------------------------------------
                printf("Exiting.\n");

                // free everything
                cap.release();
                hipFree(d_Pin);
                hipFree(d_Pout);
                free(h_Pout);

                // exit the program
                exit(0);
            }

            case 'c' : {
                // ------------------CPU CASE---------------------------------------------------------------------------
                // apply the operations
                cv::Mat frame_out(cv::Size(width, height), CV_8UC1);
                cpu_blur(frame_in, frame_out);
                // place some text on the image
                cv::putText(frame_out, "CPU Blur", cv::Point(30, 30),
                            cv::FONT_HERSHEY_COMPLEX_SMALL, 1, cv::Scalar(200, 200, 250), 1, CV_AA);

                // show the images
                cv::imshow("Source", frame_in);
                cv::imshow("Result", frame_out);
                break;
            }

            case '1' : {
                // ------------------ GPU CONVOLVE ---------------------------------------------------------------------
                // copy memory from host to device
                gpuErrchk( hipMemcpy(d_Pin, frame_in.data, pixels * sizeof(unsigned char), hipMemcpyHostToDevice) );

                // execute gpu kernel
                gpu_convolve2d <<< numBlocks, numThreads >>> (d_Pout, d_Pin, width, height);
                gpuErrchk( hipPeekAtLastError() );

                // make sure all the threads finished
                gpuErrchk( hipDeviceSynchronize() );

                // copy memory back to host from device
                gpuErrchk( hipMemcpy(h_Pout, d_Pout, sizeof(unsigned char) * pixels, hipMemcpyDeviceToHost) );

                // create container for new image and insert the data
                cv::Mat frame_out(cv::Size(width, height), CV_8UC1, h_Pout);

                // place some text on the image
                cv::putText(frame_out, "GPU Convolve", cv::Point(30, 30),
                            cv::FONT_HERSHEY_COMPLEX_SMALL, 1, cv::Scalar(200, 200, 250), 1, CV_AA);

                // show the images
                cv::imshow("Source", frame_in);
                cv::imshow("Result", frame_out);
                break;
            }

            case '2' : {
                // ------------------ GPU Naive Blur -------------------------------------------------------------------
                // copy memory from host to device
                gpuErrchk( hipMemcpy(d_Pin, frame_in.data, pixels * sizeof(unsigned char), hipMemcpyHostToDevice) );

                // execute gpu kernel
                gpu_blur <<< numBlocks, numThreads >>> (d_Pout, d_Pin, width, height);

                gpuErrchk( hipGetLastError() );

                // make sure all the threads finished
                gpuErrchk( hipDeviceSynchronize() );

                // copy memory back to host from device
                gpuErrchk( hipMemcpy(h_Pout, d_Pout, sizeof(unsigned char) * pixels, hipMemcpyDeviceToHost) );

                // create container for new image and insert the data
                cv::Mat frame_out(cv::Size(width, height), CV_8UC1, h_Pout);

                // place some text on the image
                cv::putText(frame_out, "GPU Naive Blur", cv::Point(30, 30),
                            cv::FONT_HERSHEY_COMPLEX_SMALL, 1, cv::Scalar(200, 200, 250), 1, CV_AA);

                // show the images
                cv::imshow("Source", frame_in);
                cv::imshow("Result", frame_out);
                break;
            }

            case '3' : {
                // ------------------ GPU Naive Blur -------------------------------------------------------------------
                // copy memory from host to device
                gpuErrchk( hipMemcpy(d_Pin, frame_in.data, pixels * sizeof(unsigned char), hipMemcpyHostToDevice) );

                // execute gpu kernel
                gpu_test <<< numBlocks, numThreads >>> (d_Pout, d_Pin, width, height);

                gpuErrchk( hipGetLastError() );

                // make sure all the threads finished
                gpuErrchk( hipDeviceSynchronize() );

                // copy memory back to host from device
                gpuErrchk( hipMemcpy(h_Pout, d_Pout, sizeof(unsigned char) * pixels, hipMemcpyDeviceToHost) );

                // create container for new image and insert the data
                cv::Mat frame_out(cv::Size(width, height), CV_8UC1, h_Pout);

                // place some text on the image
                cv::putText(frame_out, "GPU Test", cv::Point(30, 30),
                            cv::FONT_HERSHEY_COMPLEX_SMALL, 1, cv::Scalar(200, 200, 250), 1, CV_AA);

                // show the images
                cv::imshow("Source", frame_in);
                cv::imshow("Result", frame_out);
                break;
            }

            default: {
                cv::Mat frame_out(cv::Size(width, height), CV_8UC1, h_Pout);
                cv::putText(frame_out, "INCORRECT KEYSTROKE", cv::Point(height/2, width/2),
                            cv::FONT_HERSHEY_COMPLEX_SMALL, 3, cv::Scalar(200, 200, 250), 1, CV_AA);
                // show the images
                cv::imshow("Source", frame_in);
                cv::imshow("Result", frame_out);
                break;
            }
        }
    }
}